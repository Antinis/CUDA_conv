#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <random>
#include <time.h>

const int alignment = 32; // 32 byte alignment
const int size = 100;
const int kernel = 3;  // odd
const int batch_size = 128;
const int in_channel = 128;
const int out_channel = 128;

#define InitRandom()                         \
  std::random_device r;                      \
  std::default_random_engine generator; \
  std::uniform_int_distribution<> distribution(0, 255);

#define a(_n, _x, _y, _c) a[(_n) * size * size * in_channel + (_x) * size * in_channel + (_y) * in_channel + (_c)]
#define w(_x, _y, _ci, _co) w[(_x) * kernel * in_channel * out_channel + (_y) * in_channel * out_channel + (_ci) * out_channel + (_co)]
#define b(_n, _x, _y, _c) b[(_n) * size * size * out_channel + (_x) * size * out_channel + (_y) * out_channel + (_c)]

/// \brief Generate [N, H, W, C] input tensor and [H, W, I, O] kernel tensor.
void Generate(uint8_t *const a, uint8_t *const w) {
#pragma omp parallel for
  // Batch dimension.
  for (int s = 0; s < batch_size; ++s) {
      InitRandom();
      // Height dimension.
      for (int i = 0; i < size; ++i)
        // Width dimension.
        for (int j = 0; j < size; ++j) {
          const int channel_lower = s * size * size * in_channel
                                  + i * size * in_channel
                                  + j * in_channel;
          const int channel_upper = channel_lower + in_channel; 
          // Channel dimension.
          for (int c = channel_lower; c < channel_upper; ++c)
            a[c] = distribution(generator);
        }
  }
#pragma omp parallel for
  for (int i = 0; i < kernel; ++i) {
    InitRandom();
    for (int j = 0; j < kernel; ++j) 
      for (int CI = 0; CI < in_channel; ++CI) {
        const int channel_lower = i * kernel * in_channel * out_channel
                                + j * in_channel * out_channel
                                + CI * out_channel;
        const int channel_upper = channel_lower + out_channel;
        for (int CO = channel_lower; CO < channel_upper; ++CO) 
          w[CO] = distribution(generator);
      }
  }
}

void conv2d_cpu_kernel(const uint8_t *__restrict__ a, 
                       const uint8_t *__restrict__ w, 
                       uint8_t *__restrict__ b) {
#pragma omp parallel for
  for (int s = 0; s < batch_size; ++s) {
    size_t output_bytes = ((out_channel * sizeof(uint8_t)) + (size_t)alignment - 1) & ~((size_t)alignment -1); 
    uint8_t *packedB = static_cast<uint8_t *>(malloc(output_bytes));

    size_t input_bytes = ((kernel * kernel * in_channel * sizeof(uint8_t)) + (size_t)alignment - 1) & ~((size_t)alignment - 1);
    uint8_t *packedA = static_cast<uint8_t *>(malloc(input_bytes));

    for (int i = 0; i < size; ++i)
      for (int j = 0; j < size; ++j) {
        // Collected needed input data,
        // Start from A[s, i - kernel / 2, j - kernel / 2, 0].
        int x = i - kernel / 2;
        int y = j - kernel / 2;
        int input_index = s * size * size * in_channel
                        + x * size * in_channel
                        + y * in_channel;
        memset(packedA, 0, input_bytes);
        int A_buffer_index = 0;
        for (int kh = 0; kh < kernel; ++kh) {
          for (int kw = 0; kw < kernel; ++ kw) {
            if (!(x < 0 || x >= size || y < 0 || y >= size)) {
              memcpy(packedA + A_buffer_index, a + input_index, in_channel * sizeof(uint8_t));
            }
            else {
              memset(packedA + A_buffer_index, 0, in_channel * sizeof(uint8_t));
            }
            y++;
            A_buffer_index += in_channel;
            input_index += in_channel;
          }
          x++;
          y -= kernel;
          input_index = input_index - kernel * in_channel + size * in_channel;
        }

        // Start from B[s, i, j, 0]
        int output_index = s * size * size * out_channel 
                         + i * size * out_channel 
                         + j * out_channel;                 
        memset(packedB, 0, output_bytes);

        // Start from W[0, 0, 0, 0]
        int kernel_index = 0;
        A_buffer_index = 0;
        // Convolution 2D computation.
        // iterate over each in_channel of input tensor,
        // and accumulate contribution to output tensor.
        for (int N = 0; N < kernel * kernel; ++N) {
          for (int CI = 0; CI < in_channel; ++CI) {
            for (int CO = 0; CO < out_channel; ++CO) {
              packedB[CO] +=  packedA[A_buffer_index] * w[kernel_index];
              kernel_index++; // move to next output channel.
            }
            A_buffer_index++;
          }
        }
        memcpy(b + output_index, packedB, sizeof(uint8_t) * out_channel);
      }
    free(packedA);
    free(packedB);
  }
}

time_t Check(const uint8_t *const a, const uint8_t *const w, uint8_t *const b) {
  auto b_std = new uint8_t[batch_size * size * size * out_channel];

  std::cout << "Conv2d CPU Kernel Start... \n";
  time_t cpu_time_start, cpu_time_end;
  cpu_time_start=clock();
  conv2d_cpu_kernel(a, w, b_std);
  cpu_time_end=clock();
  std::cout << "Checking Results... \n";
  time_t cpu_time=(double)((cpu_time_end-cpu_time_start)/CLOCKS_PER_SEC);

  size_t N = batch_size * size * size * out_channel;
  for (size_t i = 0; i < N; ++i) {
    if (b[i] != b_std[i]) {
      std::cout << "\x1b[31m"
                   "Wrong Answer"
                   "\x1b[0m"
                   " at "
                << i << std::endl;
      std::cout << "expected " << (int)b_std[i] << " but found " << (int)b[i]
                << std::endl;
      delete[] b_std;
      return cpu_time;
    }
  }
  std::cout << "\x1b[32m"
               "Correct"
               "\x1b[0m"
            << std::endl;

  delete[] b_std;

  return cpu_time;
}

/// \brief Do Conv2d with NHWC Input with HWIO Kernel, and NHWC output 
__global__ void conv2d_cuda_kernel(const uint8_t *a, 
                                   const uint8_t *w, 
                                   uint8_t *b) 
{
    //i是横向编号
    //j是纵向编号
    const int batch_id=blockIdx.x;  //不会变
    const int x=blockIdx.y;         //不会变
    const int y=blockIdx.z;         //不会变
    const int out_channel_id=threadIdx.x;   //只有这个会变

    //遍历卷积核每一个像素点
    uint8_t conv=0;
    for(int in_channel_id=0; in_channel_id<in_channel; in_channel_id++)
    {
        int kx = x - kernel / 2;
        int ky = y - kernel / 2; //待卷积位置的左上角坐标
        for(int k=0; k<kernel; k++)
        {
            for(int l=0; l<kernel; l++)
            {
                if(!(kx<0 || kx>=size || ky<0 || ky>=size))
                    conv+=a(batch_id, kx, ky, in_channel_id)*w(k, l, in_channel_id, out_channel_id);
                ky++;
            }
            kx++;
            ky-=kernel;
        }
    }
    // Write back to b.
    b(batch_id, x, y, out_channel_id) = conv;

    return;
}

// naive and shit
// only for testing correctness and precision
void conv_cuda(const uint8_t *const a, const uint8_t *const w, uint8_t *const b,
               hipEvent_t *start_e, hipEvent_t *stop_e) 
{
    //建立显存上的存储空间并拷贝变量
    uint8_t *a_kernel, *w_kernel, *b_kernel;
    //size是长宽尺寸，是正方形
    hipMalloc(&a_kernel, batch_size * size * size * in_channel * sizeof(uint8_t));
    hipMemcpy(a_kernel, a, batch_size * size * size * in_channel * sizeof(uint8_t),
                hipMemcpyHostToDevice);
    hipMalloc(&w_kernel, kernel * kernel * in_channel * out_channel * sizeof(uint8_t));
    hipMemcpy(w_kernel, w, kernel * kernel * in_channel * out_channel * sizeof(uint8_t),
                hipMemcpyHostToDevice);
    hipMalloc(&b_kernel, batch_size * size * size * out_channel * sizeof(uint8_t));
    // Start Timer.
    hipEventRecord(*start_e);
    // Run Conv2d Kernel,
    // Timer for computation cuda kernel.
    //规定计算器件的数量和维度尺寸（三维）,在该例中第三维（长度维度）自动补1
    //gird是高宽都是100个盒子（size），长度方向盒子数1的区域
    //block是高宽都是16个线程（block_size），长度方向线程数1的盒子

    //这一版并行化了batch、CI和CO维度

    dim3 grid(batch_size, size, size);  //盒子间并行化
    dim3 block(out_channel); //10个盒子并行一张图片的处理
    //负责计算的核函数
    conv2d_cuda_kernel<<<grid, block>>>(a_kernel, w_kernel, b_kernel);
    hipDeviceSynchronize();
    // Stop Timer
    hipEventRecord(*stop_e);
    hipEventSynchronize(*stop_e);

    hipMemcpy(b, b_kernel, batch_size * size * size * out_channel * sizeof(uint8_t),
                hipMemcpyDeviceToHost);
    hipFree(a_kernel);
    hipFree(w_kernel);
    hipFree(b_kernel);
}

int main() {
    //a是原图像张量，w是卷积核张量，b是卷积结果张量
    auto a = new uint8_t[batch_size * size * size * in_channel];
    auto w = new uint8_t[kernel * kernel * in_channel * out_channel];
    auto b = new uint8_t[batch_size * size * size * out_channel];
    //随机生成a和w
    std::cout << "Generating input and kernel tensor... \n";
    Generate(a, w);

    //记录时间的变量
    hipEvent_t start_e, stop_e;
    hipEventCreate(&start_e);
    hipEventCreate(&stop_e);

    // Conv(a, w, b)，用CUDA;
    std::cout << "Conv2d Cuda Kernel Start... \n";
    conv_cuda(a, w, b, &start_e, &stop_e);

    //检查正确性，check函数里使用CPU循环操作卷积
    std::cout << "Verifying... \n";
    time_t cpu_time=Check(a, w, b);
    //CUDA计算时间，毫秒
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_e, stop_e);
    std::cout <<"CUDA time: " << milliseconds << " ms" << std::endl;
    std::cout << "accelerate ratio: " << cpu_time*1000/milliseconds << std::endl;
    //销毁变量 
    hipEventDestroy(start_e);
    hipEventDestroy(stop_e);

    // Output(a, w, b);
    delete[] a;
    delete[] w;
    delete[] b;
    return 0;
}