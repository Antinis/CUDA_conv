#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <random>
#include <time.h>
#include <unistd.h>
#include "hipblas.h"

const int alignment = 32; // 32 byte alignment
const int size = 100;
const int kernel = 3;  // odd
const int batch_size = 128;
const int in_channel = 128;
const int out_channel = 128;

const int block_size_m=128;
const int block_size_n=128;
const int block_size_k=8;
const int thread_size_x=16;
const int thread_size_y=16;

#define InitRandom()                         \
  std::random_device r;                      \
  std::default_random_engine generator(r()); \
  std::uniform_int_distribution<> distribution(0, 255);

#define a(_n, _x, _y, _c) a[(_n) * size * size * in_channel + (_x) * size * in_channel + (_y) * in_channel + (_c)]
#define w(_x, _y, _ci, _co) w[(_x) * kernel * in_channel * out_channel + (_y) * in_channel * out_channel + (_ci) * out_channel + (_co)]
#define b(_n, _x, _y, _c) b[(_n) * size * size * out_channel + (_x) * size * out_channel + (_y) * out_channel + (_c)]

/// \brief Generate [N, H, W, C] input tensor and [H, W, I, O] kernel tensor.
void Generate(uint8_t *const a, uint8_t *const w) {
#pragma omp parallel for
  // Batch dimension.
  for (int s = 0; s < batch_size; ++s) {
      InitRandom();
      // Height dimension.
      for (int i = 0; i < size; ++i)
        // Width dimension.
        for (int j = 0; j < size; ++j) {
          const int channel_lower = s * size * size * in_channel
                                  + i * size * in_channel
                                  + j * in_channel;
          const int channel_upper = channel_lower + in_channel; 
          // Channel dimension.
          for (int c = channel_lower; c < channel_upper; ++c)
            a[c] = distribution(generator);
        }
  }
#pragma omp parallel for
  for (int i = 0; i < kernel; ++i) {
    InitRandom();
    for (int j = 0; j < kernel; ++j) 
      for (int CI = 0; CI < in_channel; ++CI) {
        const int channel_lower = i * kernel * in_channel * out_channel
                                + j * in_channel * out_channel
                                + CI * out_channel;
        const int channel_upper = channel_lower + out_channel;
        for (int CO = channel_lower; CO < channel_upper; ++CO) 
          w[CO] = distribution(generator);
      }
  }
}

void conv2d_cpu_kernel(const uint8_t *__restrict__ a, 
                       const uint8_t *__restrict__ w, 
                       uint8_t *__restrict__ b) {
#pragma omp parallel for
  for (int s = 0; s < batch_size; ++s) {
    size_t output_bytes = ((out_channel * sizeof(uint8_t)) + (size_t)alignment - 1) & ~((size_t)alignment -1); 
    uint8_t *packedB = static_cast<uint8_t *>(malloc(output_bytes));

    size_t input_bytes = ((kernel * kernel * in_channel * sizeof(uint8_t)) + (size_t)alignment - 1) & ~((size_t)alignment - 1);
    uint8_t *packedA = static_cast<uint8_t *>(malloc(input_bytes));

    for (int i = 0; i < size; ++i)
      for (int j = 0; j < size; ++j) {
        // Collected needed input data,
        // Start from A[s, i - kernel / 2, j - kernel / 2, 0].
        int x = i - kernel / 2;
        int y = j - kernel / 2;
        int input_index = s * size * size * in_channel
                        + x * size * in_channel
                        + y * in_channel;
        memset(packedA, 0, input_bytes);
        int A_buffer_index = 0;
        for (int kh = 0; kh < kernel; ++kh) {
          for (int kw = 0; kw < kernel; ++ kw) {
            if (!(x < 0 || x >= size || y < 0 || y >= size)) {
              memcpy(packedA + A_buffer_index, a + input_index, in_channel * sizeof(uint8_t));
            }
            else {
              memset(packedA + A_buffer_index, 0, in_channel * sizeof(uint8_t));
            }
            y++;
            A_buffer_index += in_channel;
            input_index += in_channel;
          }
          x++;
          y -= kernel;
          input_index = input_index - kernel * in_channel + size * in_channel;
        }

        // Start from B[s, i, j, 0]
        int output_index = s * size * size * out_channel 
                         + i * size * out_channel 
                         + j * out_channel;                 
        memset(packedB, 0, output_bytes);

        // Start from W[0, 0, 0, 0]
        int kernel_index = 0;
        A_buffer_index = 0;
        // Convolution 2D computation.
        // iterate over each in_channel of input tensor,
        // and accumulate contribution to output tensor.
        for (int N = 0; N < kernel * kernel; ++N) {
          for (int CI = 0; CI < in_channel; ++CI) {
            for (int CO = 0; CO < out_channel; ++CO) {
              packedB[CO] +=  packedA[A_buffer_index] * w[kernel_index];
              kernel_index++; // move to next output channel.
            }
            A_buffer_index++;
          }
        }
        memcpy(b + output_index, packedB, sizeof(uint8_t) * out_channel);
      }
    free(packedA);
    free(packedB);
  }
}

time_t Check(const uint8_t *const a, const uint8_t *const w, uint8_t *const b) {
  auto b_std = new uint8_t[batch_size * size * size * out_channel];

  std::cout << "Conv2d CPU Kernel Start... \n";
  time_t cpu_time_start, cpu_time_end;
  cpu_time_start=clock();
  conv2d_cpu_kernel(a, w, b_std);
  cpu_time_end=clock();
  std::cout << "Checking Results... \n";
  time_t cpu_time=(double)((cpu_time_end-cpu_time_start)/CLOCKS_PER_SEC);

  size_t N = batch_size * size * size * out_channel;
  for (size_t i = 0; i < N; ++i) {
    if (b[i] != b_std[i]) {
      std::cout << "\x1b[31m"
                   "Wrong Answer"
                   "\x1b[0m"
                   " at "
                << i << std::endl;
      std::cout << "expected " << (int)b_std[i] << " but found " << (int)b[i]
                << std::endl;
      delete[] b_std;
      return cpu_time;
    }
  }
  std::cout << "\x1b[32m"
               "Correct"
               "\x1b[0m"
            << std::endl;

  delete[] b_std;

  return cpu_time;
}

//用来拷贝数据a到平铺矩阵a_mat的核函数
__global__ void cuda_im2col_a_kernel(
    uint8_t *a_mat,
    const uint8_t *a,
    int offset_co_batch,
    int offset_x,
    int offset_co_y,
    int offset_co_in_channel)
{
    int batch_id=blockIdx.x;
    int x=blockIdx.y;
    int y=blockIdx.z;
    int in_channel_id=threadIdx.x;

    //当前核函数处理的a_mat元素的指针偏移量
    int a_mat_offset=batch_id*offset_co_batch+x*offset_x+y*offset_co_y+in_channel_id*offset_co_in_channel;
    
    for(int kernel_id=0; kernel_id<offset_co_in_channel; kernel_id++)
    {
        int x_offset=kernel_id/3-1;
        int y_offset=kernel_id%3-1;
        int a_x=x+x_offset;
        int a_y=y+y_offset;
        a_mat[a_mat_offset+kernel_id]=(!(a_x<0 || a_x>=size || a_y<0 || a_y>=size)) ? a(batch_id, a_x, a_y, in_channel_id):0;
    }

    return;
}

__global__ void cuda_im2col_w_kernel(uint8_t *w_mat, const uint8_t *w)
{
    int out_channel_id=blockIdx.x;
    int in_channel_id=threadIdx.x;

    //本核函数指针初始偏移量
    int w_mat_offset=kernel*kernel*out_channel*in_channel_id+out_channel_id;
    for(int kernel_id=0; kernel_id<kernel*kernel; kernel_id++)
    {
        int x=kernel_id/3;
        int y=kernel_id%3;
        w_mat[w_mat_offset+kernel_id*out_channel]=w(x, y, in_channel_id, out_channel_id);
    }

    return;
}

__global__ void cuda_GEMM_kernel(
    uint8_t *a_mat,
    uint8_t *w_mat,
    uint8_t *b_mat
)
{
    const int block_n=blockIdx.x;
    const int block_m=blockIdx.y;   //1
    const int thread_x=threadIdx.x;
    const int thread_y=threadIdx.y;

    const int m=batch_size*size*size;
    const int n=out_channel;
    const int k=kernel*kernel*in_channel;

    const int thread_x_per_block=block_size_n/thread_size_x;
    const int thread_y_per_block=block_size_m/thread_size_y;
    const int thread_num_per_block=thread_x_per_block*thread_y_per_block;
    const int thread_id=thread_x_per_block*thread_y+thread_x;

    __shared__ uint8_t As[2][block_size_k][block_size_m]; //转置过的
    __shared__ uint8_t Ms[2][block_size_k][block_size_n]; //没有转置过

    uint8_t accum[thread_size_y][thread_size_x]={0};

    //线程用寄存器
    uint8_t A_rag[2][thread_size_y];
    uint8_t M_rag[2][thread_size_x];

    const int cycles_to_load_shared_memory_a=block_size_m*block_size_k/(thread_num_per_block*8);  //1 
    const int cycles_to_load_shared_memory_m=block_size_n*block_size_k/(thread_num_per_block*8);

    //每个线程载入shared memory的暂存寄存器，容量是shared memory的容量的1/线程数倍
    uint8_t load_reg_a[8*cycles_to_load_shared_memory_a];
    uint8_t load_reg_m[8*cycles_to_load_shared_memory_m];

    const int thread_used_to_load_shared_memory_per_rol_a=block_size_k/4; //A块的每行用几个线程去读取
    const int thread_used_to_load_shared_memory_per_rol_m=block_size_n/4; //M块的每行用几个线程去读取

    const int A_rol_id_this_thread=thread_id/thread_used_to_load_shared_memory_per_rol_a;
    const int A_col_id_this_thread=thread_id%thread_used_to_load_shared_memory_per_rol_a*8;
    const int W_rol_id_this_thread=thread_id/thread_used_to_load_shared_memory_per_rol_m;
    const int W_col_id_this_thread=thread_id%thread_used_to_load_shared_memory_per_rol_m*8;

    const int rol_stride_to_load_a=thread_num_per_block/thread_used_to_load_shared_memory_per_rol_a;
    const int rol_stride_to_load_w=thread_num_per_block/thread_used_to_load_shared_memory_per_rol_m;

    //第一次预读取
    //读A
    for(int i=0; i<block_size_m; i+=rol_stride_to_load_a)
    {
        int tmp_reg_index=i/rol_stride_to_load_a*8;
        reinterpret_cast<float2 *>(&load_reg_a[tmp_reg_index])[0]=reinterpret_cast<float2 *>(&a_mat[k*(block_size_m*block_m+i+A_rol_id_this_thread)+A_col_id_this_thread])[0];
        As[0][]
    }







    return;
}

__global__ void cuda_col2im_b_kernel(const uint8_t *b_mat, uint8_t *b)
{
    int x=blockIdx.y;
    int y=blockIdx.z;
    int batch_id=blockIdx.x;

    int b_mat_offset=(size*size*batch_id+size*x+y)*out_channel;
    for(int i=0; i<out_channel; i++)
        b(batch_id, x, y, i)=b_mat[b_mat_offset+i];
    
    return;
}

void conv_cuda(const uint8_t *const a, const uint8_t *const w, uint8_t *const b,
               hipEvent_t *start_e, hipEvent_t *stop_e) 
{
    //建立显存上的存储空间并拷贝变量
    uint8_t *a_kernel, *w_kernel, *b_kernel;
    //size是长宽尺寸，是正方形
    hipMalloc(&a_kernel, batch_size * size * size * in_channel * sizeof(uint8_t));
    hipMemcpy(a_kernel, a, batch_size * size * size * in_channel * sizeof(uint8_t),
                hipMemcpyHostToDevice);
    hipMalloc(&w_kernel, kernel * kernel * in_channel * out_channel * sizeof(uint8_t));
    hipMemcpy(w_kernel, w, kernel * kernel * in_channel * out_channel * sizeof(uint8_t),
                hipMemcpyHostToDevice);
    hipMalloc(&b_kernel, batch_size * size * size * out_channel * sizeof(uint8_t));
    // Start Timer.
    hipEventRecord(*start_e);
    
    /*
      im2col+GEMM算法卷积加速
    */

    //创建平铺矩阵
    //计算展开a的尺寸（是一个矩阵）
    int a_mat_width=kernel*kernel*in_channel;
    int a_mat_height=size*size*batch_size;
    //计算展开w的尺寸（是另一个矩阵）
    int w_mat_width=out_channel;
    int w_mat_height=kernel*kernel*in_channel;

    //申请矩阵化变量空间
    uint8_t *a_mat, *w_mat;
    hipMalloc(&a_mat, a_mat_height*a_mat_width*sizeof(uint8_t));
    hipMalloc(&w_mat, w_mat_height*w_mat_width*sizeof(uint8_t));

    //计算偏移系数，防止核函数内重复计算
    int offset_co_batch=kernel*kernel*in_channel*size*size;
    int offset_co_x=kernel*kernel*in_channel*size;
    int offset_co_y=kernel*kernel*in_channel;
    int offset_co_in_channel=kernel*kernel;

    //多线程复制数据a到矩阵
    dim3 grid_a(batch_size, size, size);
    dim3 block_a(in_channel);
    
    cuda_im2col_a_kernel<<<grid_a, block_a>>>(a_mat, a_kernel, offset_co_batch, offset_co_x, offset_co_y, offset_co_in_channel);
    hipDeviceSynchronize();
    hipFree(a_kernel);

    //多线程复制数据w到矩阵mat_w，为了满足运算时取出的数据在内存中的连续性，这里的矩阵是转置的
    cuda_im2col_w_kernel<<<out_channel, in_channel>>>(w_mat, w_kernel);
    hipDeviceSynchronize();
    hipFree(w_kernel);

    //申请计算结果平铺矩阵
    int b_mat_height=size*size*batch_size;
    int b_mat_width=out_channel;
    uint8_t *b_mat;
    hipMalloc(&b_mat, b_mat_height*b_mat_width*sizeof(uint8_t));

    //a的尺寸：batch_size*size*size(1280000) kernel*kernel*in_channel(1152)
    //w的尺寸：kernel*kernel*in_channel(1152) out_channel(128)
    //b的尺寸：1280000*128
    dim3 grid_GEMM(batch_size*size*size/block_size_m, out_channel/block_size_n);
    dim3 block_GEMM(block_size_n/thread_size_x, block_size_m/thread_size_y);
    cuda_GEMM_kernel<<<grid_GEMM, block_GEMM>>>(a_mat, w_mat, b_mat);
    hipDeviceSynchronize();
    hipFree(a_mat);
    hipFree(w_mat);

    dim3 grid_b(batch_size, size, size);
    // dim3 block_b(batch_size);
    cuda_col2im_b_kernel<<<grid_b, 1>>>(b_mat, b_kernel);
    hipDeviceSynchronize();

    // Stop Timer
    hipEventRecord(*stop_e);
    hipEventSynchronize(*stop_e);

    hipMemcpy(b, b_kernel, batch_size * size * size * out_channel * sizeof(uint8_t),
                hipMemcpyDeviceToHost);
    hipFree(a_kernel);
    hipFree(w_kernel);
    hipFree(b_kernel);
}

int main() {
    //a是原图像张量，w是卷积核张量，b是卷积结果张量
    auto a = new uint8_t[batch_size * size * size * in_channel];
    auto w = new uint8_t[kernel * kernel * in_channel * out_channel];
    auto b = new uint8_t[batch_size * size * size * out_channel];
    //随机生成a和w
    std::cout << "Generating input and kernel tensor... \n";
    Generate(a, w);

    //记录时间的变量
    hipEvent_t start_e, stop_e;
    hipEventCreate(&start_e);
    hipEventCreate(&stop_e);

    // Conv(a, w, b)，用CUDA;
    std::cout << "Conv2d Cuda Kernel Start... \n";
    conv_cuda(a, w, b, &start_e, &stop_e);

    //检查正确性，check函数里使用CPU循环操作卷积
    std::cout << "Verifying... \n";
    time_t cpu_time=Check(a, w, b);
    // time_t cpu_time=1;
    //CUDA计算时间，毫秒
    float milliseconds = 1;
    hipEventElapsedTime(&milliseconds, start_e, stop_e);
    std::cout <<"CUDA time: " << milliseconds << " ms" << std::endl;
    std::cout << "accelerate ratio: " << cpu_time*1000/milliseconds << std::endl;
    //销毁变量 
    hipEventDestroy(start_e);
    hipEventDestroy(stop_e);

    // Output(a, w, b);
    delete[] a;
    delete[] w;
    delete[] b;
    return 0;
}